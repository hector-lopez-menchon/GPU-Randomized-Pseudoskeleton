#include "hip/hip_runtime.h"


#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include <hipsolver.h>
#include <assert.h>
#include <string>
#include <hip/hip_runtime.h>


#include <complex.h> //OJO

extern "C" int multiplicar_por_dos(hipDoubleComplex *A, int M, int N){
	//A: matrix
	//M: number of rows
	//N: number of cols
	for (int ii=0; ii<N; ii++){
		for (int jj=0; jj<M; jj++){
			A[ii*M+jj] = hipCmul(make_hipDoubleComplex(2.0,0.0),A[ii*M+jj]);
		}
	}
	return 1;
}


extern "C" int cuda_copy(hipDoubleComplex *A, hipDoubleComplex *B, int M, int N){
	//Para probar a copiar desde la GPU
	
	hipDoubleComplex *d_A = NULL;
	hipMalloc(&d_A,sizeof(hipDoubleComplex)*M*N);
	hipMemcpy(d_A,A,sizeof(hipDoubleComplex)*M*N,hipMemcpyHostToDevice);
	hipMemcpy(B,d_A,sizeof(hipDoubleComplex)*M*N,hipMemcpyDeviceToHost);
	
	return 1;	

}


double compute_norm(hipDoubleComplex *W, int lengthW){
	        double aux = 0.0;
		        for (int ii=0; ii<lengthW; ii++){
				                //aux = aux + pow(cabs(W[ii]),2);
				                aux = aux + pow(hipCabs(W[ii]),2);
						        }
			        return sqrt(aux);
}


void substract(hipDoubleComplex *vec, hipDoubleComplex *a, hipDoubleComplex *b, int N){
	        //Performs the operation v = a - b,
	        //vec: where the result is stored
	        //a: first vector
	        //b: second vector
	        //N: length of a and b
	        for (int ii=0; ii<N; ii++){
			                //vec[ii] = a[ii] - b[ii];
			                vec[ii] = hipCsub(a[ii],b[ii]);
					        }
		        return;
}



__global__ void kernel_substract(hipDoubleComplex *vec, hipDoubleComplex *a, hipDoubleComplex *b, int N){
        int ii = blockIdx.x*blockDim.x + threadIdx.x;
        vec[ii]=hipCsub(a[ii],b[ii]); //a[ii]-b[ii];
        return;
}

void d_substract(hipDoubleComplex *vec, hipDoubleComplex *a, hipDoubleComplex *b, int N){
        //Performs the operation v = a - b,
        //vec: where the result is stored
        //a: first vector
        //b: second vector
        //N: length of a and b

        kernel_substract<<<N,1>>>(vec,a,b,N);

	return;
}

//cuda_svd2 (basado en cuda_pruebas,que a pesar de su nombre es la versión correcta de SVD. En esta segunda función, vamos a procurar que nos devuelva las matrices en formato reducido)
extern "C" int cuda_svd2(hipDoubleComplex *U, double *S, hipDoubleComplex *Vt, hipDoubleComplex *A, int M, int N){
	//Este tal y como está FUNCIONA
	//Para probar a copiar desde la GPU

	printf("Starting cuda_svd2\n");	

	hipsolverHandle_t cusolverH = NULL;
	hipblasHandle_t cublasH = NULL;
    	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
        hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	hipError_t cudaStat1 = hipSuccess;
	hipError_t cudaStat2 = hipSuccess;
	hipError_t cudaStat3 = hipSuccess;
	hipError_t cudaStat4 = hipSuccess;
	hipError_t cudaStat5 = hipSuccess;
	hipError_t cudaStat6 = hipSuccess;


	hipDoubleComplex *d_A = NULL;
	double *d_S = NULL;
	hipDoubleComplex *d_U = NULL;
	hipDoubleComplex *d_Vt = NULL;
	int *devInfo = NULL;
	hipDoubleComplex *d_work = NULL;
	double *d_rwork = NULL;

	int lwork = 0;
	int lda = M;

	//int *d_work = NULL;
	
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    	cublas_status = hipblasCreate(&cublasH);
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	int minMN = min(M,N);

	
	cudaStat1 = hipMalloc((void**)&d_A, sizeof(hipDoubleComplex)*M*N);
	cudaStat2 = hipMalloc((void**)&d_S, sizeof(double)*minMN);
	//cudaStat3 = hipMalloc((void**)&d_U, sizeof(hipDoubleComplex)*M*M);
	cudaStat3 = hipMalloc((void**)&d_U, sizeof(hipDoubleComplex)*M*N);
	//cudaStat4 = hipMalloc((void**)&d_Vt, sizeof(hipDoubleComplex)*N*N);
	cudaStat4 = hipMalloc((void**)&d_Vt, sizeof(hipDoubleComplex)*N*N);

	cudaStat5 = hipMalloc((void**)&devInfo,sizeof(int));


	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);
	assert(hipSuccess == cudaStat4);

	cudaStat1 = hipMemcpy(d_A, A, sizeof(hipDoubleComplex)*M*N, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat1);

	cusolver_status = hipsolverDnZgesvd_bufferSize(
			cusolverH,
			M,
			N,
			&lwork);
	
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
	
	cudaStat1 = hipMalloc((void**)&d_work, sizeof(hipDoubleComplex)*lwork);
	assert(hipSuccess == cudaStat1);

	
	//Step 4: compute SVD
	signed char jobu = 'S'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	
	cusolver_status = hipsolverDnZgesvd(
			cusolverH,
			jobu,
			jobvt,
			M,
			N,
			d_A,
			lda,
			d_S,
			d_U,
			lda, //ldu
			d_Vt,
			N, //ldvt (antes lda)
			d_work,
			lwork,
			d_rwork,
			devInfo);
		    
	cudaStat1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat1);

 
	cudaStat1 = hipMemcpy(U, d_U, sizeof(hipDoubleComplex)*M*N,hipMemcpyDeviceToHost);
	cudaStat2 = hipMemcpy(Vt, d_Vt, sizeof(hipDoubleComplex)*N*N,hipMemcpyDeviceToHost);
	cudaStat3 = hipMemcpy(S, d_S, sizeof(double)*minMN,hipMemcpyDeviceToHost);
   

	//free resources
	hipFree(d_A);
	hipFree(d_S);
	hipFree(d_U);
	hipFree(d_Vt);
	hipFree(devInfo); //Posible error
	hipFree(d_work);
	hipFree(d_rwork);
	hipblasDestroy(cublasH);
	hipsolverDnDestroy(cusolverH);

	return 1;	

}



__global__ void kernel_Sinv(double *d_vec, hipDoubleComplex *d_vec_complex, int N, double treshold){
	//Performs the filtering operation and transforms to hipDoubleComplex typoe
	int ii = blockIdx.x*blockDim.x + threadIdx.x;
	if (d_vec[ii]<treshold){
		d_vec_complex[ii] = make_hipDoubleComplex(0.0,0.0);
	} else {
		d_vec_complex[ii] = make_hipDoubleComplex(1/d_vec[ii],0.0);
	}
	return;
}


extern "C" int cuda_pinv(hipDoubleComplex *pinvA, hipDoubleComplex *U, double *S, hipDoubleComplex *Vt, hipDoubleComplex *A, int M, int N, double treshold){
	//Este tal y como está FUNCIONA
	//Para probar a copiar desde la GPU
	//Based on cuda_svd2. Intended to compute pinv in cuda

	//A is MxN
	//pinvA is NxM

	printf("Starting cuda_pinv\n");	

	hipsolverHandle_t cusolverH = NULL;
	hipblasHandle_t cublasH = NULL;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
        hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	hipError_t cudaStat1 = hipSuccess;
	hipError_t cudaStat2 = hipSuccess;
	hipError_t cudaStat3 = hipSuccess;
	hipError_t cudaStat4 = hipSuccess;
	hipError_t cudaStat5 = hipSuccess;
	hipError_t cudaStat6 = hipSuccess;


	hipDoubleComplex *d_A = NULL;
	double *d_S = NULL;
	hipDoubleComplex *d_U = NULL;
	hipDoubleComplex *d_Vt = NULL;
	hipDoubleComplex *d_aux_V = NULL;  //Aux for V operations
	hipDoubleComplex *d_pinvA = NULL;
	hipDoubleComplex *d_Sinv = NULL;
	int *devInfo = NULL;
	hipDoubleComplex *d_work = NULL;
	double *d_rwork = NULL;

	int lwork = 0;
	const hipDoubleComplex h_alpha = make_hipDoubleComplex(1.0,0.0);
	const hipDoubleComplex h_beta  = make_hipDoubleComplex(0.0,0.0);

	int lda = M;

	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    	cublas_status = hipblasCreate(&cublasH);
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	int minMN = min(M,N);
	
	cudaStat1 = hipMalloc((void**)&d_A, sizeof(hipDoubleComplex)*M*N);
	cudaStat2 = hipMalloc((void**)&d_S, sizeof(double)*minMN);
	cudaStat3 = hipMalloc((void**)&d_U, sizeof(hipDoubleComplex)*M*N);
	cudaStat4 = hipMalloc((void**)&d_Vt, sizeof(hipDoubleComplex)*N*N);
	cudaStat6 = hipMalloc((void**)&d_aux_V, sizeof(hipDoubleComplex)*N*N); //disorder in cudaStat 6 and 5
	cudaStat6 = hipMalloc((void**)&d_pinvA, sizeof(hipDoubleComplex)*N*M); //reusing cudaStat6
	cudaStat6 = hipMalloc((void**)&d_Sinv, sizeof(hipDoubleComplex)*N);
	cudaStat5 = hipMalloc((void**)&devInfo,sizeof(int));


	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);
	assert(hipSuccess == cudaStat4);

	cudaStat1 = hipMemcpy(d_A, A, sizeof(hipDoubleComplex)*M*N, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat1);

	cusolver_status = hipsolverDnZgesvd_bufferSize(
			cusolverH,
			M,
			N,
			&lwork);
	
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
	
	cudaStat1 = hipMalloc((void**)&d_work, sizeof(hipDoubleComplex)*lwork);
	assert(hipSuccess == cudaStat1);

	
	//Step 4: compute SVD
	signed char jobu = 'S'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	
	cusolver_status = hipsolverDnZgesvd(
			cusolverH,
			jobu,
			jobvt,
			M,
			N,
			d_A,
			lda,
			d_S,
			d_U,
			lda, //ldu
			d_Vt,
			N, //ldvt (antes lda)
			d_work,
			lwork,
			d_rwork,
			devInfo);
		    
	cudaStat1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat1);

	kernel_Sinv<<<N,1>>>(d_S, d_Sinv, N, treshold);

	//Compute d_aux_V = d_Sinv*Vt
	cublas_status = hipblasZdgmm(
			cublasH, HIPBLAS_SIDE_LEFT,
			N,N,
			d_Vt, N,
			d_Sinv,1,
			d_aux_V,N);

	//Compute (d_Sinv*Vt)'*U'=V*d_Sinv*U'
	cublas_status = hipblasZgemm(
			cublasH,
			HIPBLAS_OP_C, HIPBLAS_OP_C,
			N, M, N,
			&h_alpha,
			d_aux_V, N,
			d_U, M,
			&h_beta,
			d_pinvA,N);

	cudaStat1 = hipMemcpy(U, d_U, sizeof(hipDoubleComplex)*M*N,hipMemcpyDeviceToHost);
	cudaStat2 = hipMemcpy(Vt, d_aux_V, sizeof(hipDoubleComplex)*N*N,hipMemcpyDeviceToHost);
	cudaStat3 = hipMemcpy(S, d_S, sizeof(double)*minMN,hipMemcpyDeviceToHost); //>For debug. Originally d_SS
	cudaStat4 = hipMemcpy(pinvA, d_pinvA, sizeof(hipDoubleComplex)*N*M, hipMemcpyDeviceToHost);
   

	//free resources
	hipFree(d_A);
	hipFree(d_S);
	hipFree(d_U);
	hipFree(d_Vt);
	hipFree(d_aux_V);
	hipFree(d_pinvA);
	hipFree(d_Sinv);
	hipFree(devInfo); //Posible error
	hipFree(d_work);
	hipFree(d_rwork);
	
	hipblasDestroy(cublasH);
	hipsolverDnDestroy(cusolverH);


	printf("CAFE CON LOU\n");

	return 1;	

}




__global__ void kernel_filter(double *d_V, int N, double treshold){
	int ii = blockIdx.x*blockDim.x + threadIdx.x;
	if (d_V[ii]<treshold){
		d_V[ii]=0.0;
	} else {
		d_V[ii]=1.0/d_V[ii];
	}
	return;
}

extern "C" int cuda_filter(double *V, double treshold, int N){
	//V: vector to perform filtering
	//treshold: cut off for filtering
	//N: length of the vector

	double *d_V = NULL;

	hipMalloc(&d_V,N*sizeof(double));
	hipMemcpy(d_V,V,N*sizeof(double),hipMemcpyHostToDevice);
	
	kernel_filter<<<N,1>>>(d_V,N,treshold);

	hipMemcpy(V,d_V,N*sizeof(double),hipMemcpyDeviceToHost);

	return 1;

}

extern "C" int cuda_svd(hipDoubleComplex *U, double *S, hipDoubleComplex *Vt, hipDoubleComplex *A, int M, int N){
	//Se han hecho cambios erroneos. No confiar en este código

	//U: U matrix of the SVD
	//S: singular values of the SVD
	//V: V matrix of the SVD
	//A: matrix to be decomposed
	//M: number of rows of A
	//N: number of columns of A
	
	hipsolverHandle_t cusolverH = NULL;
    	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
        hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	hipError_t cudaStat1 = hipSuccess;
	hipError_t cudaStat2 = hipSuccess;
	hipError_t cudaStat3 = hipSuccess;
	hipError_t cudaStat4 = hipSuccess;
	hipError_t cudaStat5 = hipSuccess;
	hipError_t cudaStat6 = hipSuccess;

	hipDoubleComplex *d_A = NULL;
	double *d_S = NULL;
	hipDoubleComplex *d_U = NULL;
	hipDoubleComplex *d_Vt = NULL;
	int *devInfo = NULL;
	hipDoubleComplex *d_work = NULL;
	double *d_rwork = NULL;

	int lwork = 0;
	int lda = M;

	//int *d_work = NULL;
	
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    	//cublas_status = hipblasCreate(&cublasH);
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	int minMN = min(M,N);

	printf("M = %d\n",M);
	printf("N = %d\n",N);

      cudaStat1 = hipMalloc((void**)&d_A, sizeof(hipDoubleComplex)*M*N);
      cudaStat2 = hipMalloc((void**)&d_S, sizeof(double)*minMN);
      cudaStat3 = hipMalloc((void**)&d_U, sizeof(hipDoubleComplex)*M*M);
      cudaStat4 = hipMalloc((void**)&d_Vt, sizeof(hipDoubleComplex)*N*N);
      cudaStat5 = hipMalloc((void**)&devInfo,sizeof(int));

	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);
	assert(hipSuccess == cudaStat4);

	cudaStat1 = hipMemcpy(d_A, A, sizeof(hipDoubleComplex)*M*N, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat1);

	//step 3 query working space
	cusolver_status = hipsolverDnZgesvd_bufferSize(
			cusolverH,
			M,
			N,
			&lwork);
	
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
	
	cudaStat1 = hipMalloc((void**)&d_work, sizeof(hipDoubleComplex)*lwork);
	assert(hipSuccess == cudaStat1);

	
	//Step 4: compute SVD
	signed char jobu = 'A'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	
	cusolver_status = hipsolverDnZgesvd(
			cusolverH,
			jobu,
			jobvt,
			M,
			N,
			d_A,
			lda,
			d_S,
			d_U,
			lda, //ldu
			d_Vt,
			N, //ldvt (antes lda)
			d_work,
			lwork,
			d_rwork,
			devInfo);
	    
	cudaStat1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat1);

	cudaStat1 = hipMemcpy(U, d_U, sizeof(hipDoubleComplex)*M*M,hipMemcpyDeviceToHost);
	cudaStat2 = hipMemcpy(Vt, d_Vt, sizeof(hipDoubleComplex)*N*N,hipMemcpyDeviceToHost);
	cudaStat3 = hipMemcpy(S, d_S, sizeof(double)*minMN,hipMemcpyDeviceToHost);


	//free resources
	hipFree(d_A);
	hipFree(d_S);
	hipFree(d_U);
	hipFree(d_Vt);
	hipFree(devInfo); //Posible error
	hipFree(d_work);
	hipFree(d_rwork);
	
	//hipblasDestroy(cublasH);
	hipsolverDnDestroy(cusolverH);

	return 1;


}

int get_proper_cols(hipDoubleComplex *Asubset, hipDoubleComplex *A, int *col_vec, int M, int N, int length_col_vec){
        //Output
        //Asubset: selected cols of A

        //Input
        //A matrix to extract cols
        //col_vec: vector containing the rows to be extracted
        //M: number of rows of A
        //N: number of cols of A
        //length_col_vec: number of elements of col_vec

        for (int ii=0; ii<length_col_vec; ii++){
                for (int jj=0; jj<M; jj++){
                        Asubset[ii*M+jj] = A[(col_vec[ii]-1)*M+jj];
                }
        }
        return 1;
}


int receive_integer_vector(int *v, int N){
        //Let's assume v has length 4
        for (int ii=0;ii<4;ii++){
                printf("v[%d] = %d\n",ii,v[ii]);
        }
        return 1;
}


int get_proper_rows(hipDoubleComplex *Asubset, hipDoubleComplex *A, int *row_vec, int M, int N, int length_row_vec){
        //Output
        //Asubset: selected rows of A

        //Input
        //A matrix to rows
        //row vec: vector containing the rows to be extracted
        //M: number of rows of A
        //N: number of cols of A
        //length_row_vec: number of elements of row_vec

        for (int ii=0; ii<N; ii++){
                for (int jj=0; jj<length_row_vec; jj++){
                        Asubset[ii*length_row_vec+jj] = A[ii*M+(row_vec[jj]-1)];
                }
        }

        return 1;
}





extern "C" int aux_pinv_compression(hipDoubleComplex *C, hipDoubleComplex *U, hipDoubleComplex *R, hipDoubleComplex *A, int *row_samples, int *col_samples, int ns, int M, int N, double tolerance){
	
	//This function returns the CUR compression of A
	//C: Mxns matrix
	//U: nsxns matrix
	//R: nsxN matrix
	//A: MxN matrix to be comressed
	//row_samples: vector of length ns containing the indices of the rows to take (Matlab notation ie start at 1)
	//col_samples: idem for columns
	//ns: number of samples
	//M: number of rows of A
	//N: number of cols of A
	//tolerance: tolerance for performing pseudoinverse

	int aux = 0;
	aux = get_proper_cols(C,A,col_samples,M,N,ns);
	aux = get_proper_rows(R,A,row_samples,M,N,ns);

	

	hipDoubleComplex *Uintersection = NULL; //The interesection of C and R
	hipDoubleComplex *Usvd = NULL; //The U of the SVD
	double *Ssvd = NULL; //The S of the SVD
	hipDoubleComplex *Vtsvd = NULL; //The Vt of the SVD

	Uintersection = (hipDoubleComplex *)malloc(ns*ns*sizeof(hipDoubleComplex));
	Usvd  = (hipDoubleComplex *)malloc(ns*ns*sizeof(hipDoubleComplex));
	Ssvd  = (double *)malloc(ns*sizeof(hipDoubleComplex));
	Vtsvd =	(hipDoubleComplex *)malloc(ns*ns*sizeof(hipDoubleComplex));

	for (int ii=0; ii<ns; ii++){
		for (int jj=0; jj<ns; jj++){
			Uintersection[ii*ns+jj] = A[(col_samples[ii]-1)*M+(row_samples[jj]-1)];
		}
	}	

	cuda_pinv(U,Usvd,Ssvd,Vtsvd,Uintersection,ns,ns,tolerance);	
	return 1;
}

extern "C" double compute_error_compression(hipDoubleComplex *C, hipDoubleComplex *U, hipDoubleComplex *R, hipDoubleComplex *Avtest_old, hipDoubleComplex *Avtest_new, hipDoubleComplex *vtest, int M1, int N1, int M2){  
	
	//C is M1xN1
	//U is N1xN1
	//R is N1xM2
	//vtest is M2x1
       //Avtest_old and Avtest_new are M1x1

	hipsolverHandle_t cusolverH = NULL;
	hipblasHandle_t cublasH = NULL;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;


	cusolver_status = hipsolverDnCreate(&cusolverH);
	cublas_status = hipblasCreate(&cublasH);
 
 	const hipDoubleComplex h_alpha = make_hipDoubleComplex(1.0,0.0);
        const hipDoubleComplex h_beta  = make_hipDoubleComplex(0.0,0.0);

	double err_here = 0.0;
	
	double norm_Avtest_new = 0.0;
	double norm_Avtest_diff = 0.0;

	hipDoubleComplex *d_C = NULL;
	hipDoubleComplex *d_U = NULL;
	hipDoubleComplex *d_R = NULL;
	hipDoubleComplex *d_Avtest_old = NULL;
	hipDoubleComplex *d_Avtest_new = NULL;
	hipDoubleComplex *d_vtest = NULL;
	hipDoubleComplex *d_aux1 = NULL; //For storing the res of R*v
	hipDoubleComplex *d_aux2 = NULL; //For storing the res of U*(R*v)
	hipDoubleComplex *d_aux3 = NULL; //For storing the res of Avtest_old-Avtest_new

	hipMalloc((void**)&d_C, sizeof(hipDoubleComplex)*M1*N1);
	hipMalloc((void**)&d_U, sizeof(hipDoubleComplex)*N1*N1);
	hipMalloc((void**)&d_R, sizeof(hipDoubleComplex)*N1*M2);
	hipMalloc((void**)&d_Avtest_old, sizeof(hipDoubleComplex)*M1);
	hipMalloc((void**)&d_Avtest_new, sizeof(hipDoubleComplex)*M1);
	hipMalloc((void**)&d_vtest, sizeof(hipDoubleComplex)*M2);
	hipMalloc((void**)&d_aux1, sizeof(hipDoubleComplex)*N1);
	hipMalloc((void**)&d_aux2, sizeof(hipDoubleComplex)*N1);
	hipMalloc((void**)&d_aux3, sizeof(hipDoubleComplex)*M1);


	hipMemcpy(d_C,C, sizeof(hipDoubleComplex)*M1*N1,hipMemcpyHostToDevice);
	hipMemcpy(d_U,U, sizeof(hipDoubleComplex)*N1*N1,hipMemcpyHostToDevice);
	hipMemcpy(d_R,R, sizeof(hipDoubleComplex)*N1*M2,hipMemcpyHostToDevice);
	hipMemcpy(d_Avtest_old,Avtest_old, sizeof(hipDoubleComplex)*M1,hipMemcpyHostToDevice);
	hipMemcpy(d_Avtest_new,Avtest_new, sizeof(hipDoubleComplex)*M1,hipMemcpyHostToDevice);
	hipMemcpy(d_vtest,vtest, sizeof(hipDoubleComplex)*M2,hipMemcpyHostToDevice);

	//R*vtest
	cublas_status = hipblasZgemm(
			cublasH,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			N1, 1, M2,
			&h_alpha,
			d_R, N1,
			d_vtest, M2,
			&h_beta,
			d_aux1,N1);
	//U*d_aux1 = U*(R*vtest)
	cublas_status = hipblasZgemm(
			cublasH,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			N1,1 , N1,
			&h_alpha,
			d_U, N1,
			d_aux1, N1,
			&h_beta,
			d_aux2,N1);
	//C*d_aux12 = C*(U*(R*vtest))
	cublas_status = hipblasZgemm(
			cublasH,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			M1, 1 , N1,
			&h_alpha,
			d_C, M1,
			d_aux2, N1,
			&h_beta,
			d_Avtest_new,M1);
	hipMemcpy(Avtest_new,d_Avtest_new, sizeof(hipDoubleComplex)*M1,hipMemcpyDeviceToHost);
	
	cublas_status = hipblasDznrm2(cublasH,M1,d_Avtest_new,1,&norm_Avtest_new);
	d_substract(d_aux3,d_Avtest_old,d_Avtest_new,M1);	
	cublas_status = hipblasDznrm2(cublasH,M1,d_aux3,1,&norm_Avtest_diff);

	double norm_diff_dummy = 0.0;
	double norm_new_dummy  = 0.0;
	hipDoubleComplex *diff_dummy;
	diff_dummy = (hipDoubleComplex *)malloc(M1*sizeof(hipDoubleComplex));
	substract(diff_dummy,Avtest_old,Avtest_new,M1);
	norm_diff_dummy = compute_norm(diff_dummy,M1);
	norm_new_dummy  = compute_norm(Avtest_new,M1);

	err_here = norm_Avtest_diff/norm_Avtest_new;

	hipFree(d_C);
	hipFree(d_U);
	hipFree(d_R);
	hipFree(d_Avtest_old);
	hipFree(d_Avtest_new);
	hipFree(d_vtest);
	hipFree(d_aux1);
	hipFree(d_aux2);
	hipFree(d_aux3);
	
	hipblasDestroy(cublasH);
	hipsolverDnDestroy(cusolverH);

	return err_here;
}

